
#include <hip/hip_runtime.h>
#ifdef __cplusplus
extern "C" {
  #endif

  struct point{
    float x;
    float y;
  };
  
  
  __global__ void pi(const struct point* A, int* res, const int nbPoint, const float ray){
    const int idx = 32*blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < nbPoint-32*blockDim.x)
      #pragma unroll 16
      for (int j = 0; j < 32; j++) {
	int i = idx + blockDim.x * j;
	res[i] = (A[i].x*A[i].x + A[i].y*A[i].y <= ray);
      }
  }

  
  struct point2{
    double x;
    double y;
  };

  
__global__ void pi_double(const struct point2* A, int* res, const int nbPoint, const float ray){
    const int idx = 32*blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < nbPoint-32*blockDim.x)
      if (idx < (int)(nbPoint-32*blockDim.x))
	#pragma unroll 16
	for (int j = 0; j < 32; j++) {
	  int i = idx + blockDim.x * j;
	  res[i] = (A[i].x*A[i].x + A[i].y*A[i].y <= (double)ray);
	}
  }
  
  
  #ifdef __cplusplus
}
#endif
